#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <Camera.h>
#include <Convert.h>
#include <>
#include <Frame.h>
#include <HitResult.h>
#include <Objects.h>
#include <Ray.h>
#include <RayTracing.h>
#include <Rendering.h>
#include <TypeDefs.h>
#include <Vectors.h>
#include "../CudaLogExtension.h"
#include "../AthenaCuda.h"

#define BOX_COUNT_DEVICE_LIMIT			512
#define SPHERE_COUNT_DEVICE_LIMIT		512


// __constant__ memory
__device__ __constant__ double3 cameraParams[CameraParameter::Count];

// __constant__ memory accessors
__device__ const v3f& Camera(CameraParameter::Enum parameterId) { return (v3f&)cameraParams[parameterId]; }
__device__ const v3f* Camera() { return (v3f*)cameraParams; }


__device__ f32 GetRandom(ui32* seed0, ui32* seed1)
{
	/* hash the seeds using bitwise AND operations and bitshifts */
	*seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);
	*seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

	ui32 ires = ((*seed0) << 16) + (*seed1);

	/* use union struct to convert int to float */
	union
	{
		f32 f;
		ui32 ui;
	} res;

	res.ui = (ires & 0x007fffff) | 0x40000000;  /* bitwise AND, bitwise OR */
	return (res.f - 2.0f) / 2.0f;
}

__global__ void RayTracingKernel
(
	const Objects objects,
	v4b* colorBuffer,
	v4b* depthBuffer,
	v4b* normalBuffer,
	v4b* debugBuffer,
	v1ui64* objectIdBuffer,

	const v2ui pixelSize,
	const ui64 frameCount
)
{
	// blockIdx: <0; gridDim)
	// threadIdx: <0; blockDim)

	// pixel position in output image
	// x: left -> right
	// y: top -> bottom

	const v2ui pixel(
		pixelSize.x * (blockIdx.x * blockDim.x + threadIdx.x),
		pixelSize.y * (blockIdx.y * blockDim.y + threadIdx.y));
	const ui32 index = (pixel.y * blockDim.x * pixelSize.x * gridDim.x + pixel.x);

	// thread colors
	v2f threadColor((real)threadIdx.x / blockDim.x, (real)threadIdx.y / blockDim.y);
	// block color
	const real blockColor =
		((blockIdx.x % 2 == 0 && blockIdx.y % 2 != 0) || (blockIdx.x % 2 != 0 && blockIdx.y % 2 == 0)) ?
		((real)blockIdx.x / gridDim.x) * ((real)blockIdx.y / gridDim.y) : 0;

	//RenderingParameters params;
	//params.tracingMethod = TracingMethod::StraightForward;

	Ray primaryRay = Ray::GetPrimary(Camera(), pixel, pixelSize);

	// TODO raytracing
	RayTraceResult result;// = RayTrace(
		//objects, 
		//params, 
		//primaryRay, 
		//array_of<v3f>(), 
		//null, 
		//null, 
		//0);

	ui32 seed0 = pixel.x * (ui32)frameCount;
	ui32 seed1 = pixel.y * (ui32)frameCount;
	result.color.Set(
		REAL_TO_BYTE(GetRandom(&seed0, &seed1)),
		REAL_TO_BYTE(GetRandom(&seed0, &seed1)),
		REAL_TO_BYTE(GetRandom(&seed0, &seed1)));

	// color is average since last view update
	//frame.colorAccBuffer[frameOffset] += result.color;
	//result.color = frame.colorAccBuffer[frameOffset] / (real)frameCountSinceChange;

	// color
	colorBuffer[index] = Vector3fToVector4b(result.color);
	
	// depth
	depthBuffer[index] = v4b();
	
	// normal
	normalBuffer[index] = Vector3fToVector4b(vectors::Abs(primaryRay.direction));

	// debug
	debugBuffer[index].x = REAL_TO_BYTE(threadColor.x);
	debugBuffer[index].y = REAL_TO_BYTE(threadColor.y);
	debugBuffer[index].z = REAL_TO_BYTE(blockColor);
}

// wrapper for the __global__ call that sets up the kernel call
EXTERN_C void RunRayTracingKernel(const Objects& objects, 
	dim3 numOfBlocks, dim3 threadsPerBlock, v2ui pixelSize, AthenaCudaStorage* storage, const ui64 frameCount)
{
	// execute the kernel
	RayTracingKernel <<< numOfBlocks, threadsPerBlock >>>
	(
		objects,
		storage->buffer[FrameBuffer::Color].ptr,
		storage->buffer[FrameBuffer::Depth].ptr,
		storage->buffer[FrameBuffer::Normal].ptr,
		storage->buffer[FrameBuffer::Debug].ptr,
		storage->objectIdBuffer.ptr,
		pixelSize,
		frameCount
	);

	CHECK_CUDA_ERROR_LOG_TL(hipDeviceSynchronize());
}

#define COPY_OBJECTS_TO_DEVICE(o) \
	CHECK_CUDA_ERROR_LOG_TL(hipMemcpyToSymbol(HIP_SYMBOL(o), objects->o.array.ptr, sizeof(*o) * objects->o.array.count));

EXTERN_C void UpdateKernel(const v3f* camera)
{
	// copy camera properties to device
	CHECK_CUDA_ERROR_LOG_TL(hipMemcpyToSymbol(HIP_SYMBOL(cameraParams), camera, sizeof(*cameraParams) * CameraParameter::Count));
}
